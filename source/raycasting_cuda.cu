#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime_api.h"
#include "raycasting_kernel.h"
#include "raycasting_kernel.cuh"

namespace {
constexpr int blockX = 16;
constexpr int blockY = 16;
}

__global__ void Render(uchar4 *dst, const int imageW, const int imageH)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int pixel = y * imageW + x;

	if (x < imageW && y < imageH)
	{
		dst[pixel].x = (int)((float)x / imageW * 255);
		dst[pixel].y = (int)((float)y / imageH * 255);
		dst[pixel].z = 0;
	}
} 


void RenderScene(uchar4 *dst, const int imageW, const int imageH)
{
    dim3 threads(blockX, blockY);
    dim3 grid(iDivUp(imageW, blockX), iDivUp(imageH, blockY));

	Render<<<grid, threads>>>(dst, imageW, imageH);

    getLastCudaError("Raycasting kernel execution failed.\n");
}
